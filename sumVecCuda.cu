#include <stdio.h>
#include <hip/hip_runtime.h>

__global__
void vecAddKernel(float* A, float* B, float* C, int n)
{
  int i = blockDim.x*blockIdx.x + threadIdx.x;
  if(i<n) C[i] = A[i] + B[i];
}

void vecAdd(float* A, float* B, float* C, int n)
{
  int size = n * sizeof(float);
  float *d_A, *d_B, *d_C;

  hipMalloc((void **) &d_A, size);
  hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
  hipMalloc((void **) &d_B, size);
  hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
  
  hipMalloc((void **) &d_C, size);

  vecAddKernel<<<ceil(n/256.00), 256>>>(d_A, d_B, d_C, n);

  hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

  for(int i = 0; i < n; ++i)
    printf("C[%d] = %f\n",i,C[i]);

  hipFree(d_A); hipFree(d_B); hipFree(d_C);

}

int main()
{
  int n = 10;
  float* A, *B, *C;
  A = (float*)malloc(n*sizeof(float));
  B = (float*)malloc(n*sizeof(float));
  C = (float*)malloc(n*sizeof(float));
  for(int i = 0; i < n; ++i)
  {
    A[i]= 1;
    B[i]= 1;
    C[i]= 0;  
  }
  vecAdd(A,B,C,n); 
}
