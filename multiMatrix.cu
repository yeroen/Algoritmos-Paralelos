
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__
void matMultKernel(float *d_M, float *d_N, float *d_P, int Width){
  int Row = blockIdx.y*blockDim.y + threadIdx.y;
  int Col = blockIdx.x*blockDim.x + threadIdx.x;
  int k = 0;
  if(Row < Width && Col < Width){
      float Pvalue = 0;
      for(k = 0; k < Width; ++k){
          Pvalue += d_M[Row*Width + k] * d_N[k*Width+Col];
      }
      d_P[Row*Width+Col] = Pvalue;
  }
}

void matMult(float* A, float* B, float* C, int n){
  int size = n*n*sizeof(float);
  float *d_A, *d_B, *d_C;

  hipMalloc((void **) &d_A, size);
  hipMemcpy(d_A,A,size,hipMemcpyHostToDevice);
  hipMalloc((void **) &d_B, size);
  hipMemcpy(d_B,B,size,hipMemcpyHostToDevice);
  hipMalloc((void **) &d_C, size);

  dim3 dimGrid(ceil(n/8.0),ceil(n/16.0),1);
  dim3 dimBlock(8,16,1);
  matMultKernel<<<dimGrid, dimBlock>>>(d_A,d_B,d_C,n);
  
  hipMemcpy(C,d_C,size,hipMemcpyDeviceToHost);

  hipFree(d_A); hipFree(d_B); hipFree(d_C);
}


int main(){
  int n,i,j;
  float *h_A,*h_B,*h_C;
  scanf("%d", &n);
  h_A = (float*) malloc(n*n*sizeof(float));
  h_B = (float*) malloc(n*n*sizeof(float));
  h_C = (float*) malloc(n*n*sizeof(float));
  for(i = 0; i < n; i++){
    //scanf("%f", &h_A[i]);
    for(j = 0; j < n; j++)
      h_A[i*n+j] = 1;
  }
  for(i = 0; i < n; i++){
    //scanf("%f", &h_B[i]);
    for(j = 0; j < n; j++)
      h_B[i*n+j] = 1;
  }
  matMult(h_A,h_B,h_C,n);
  for(i = 0; i < n; i++){
    for(j = 0; j < n; j++){
        printf("%f ", h_C[i*n+j]);
    }
    printf("\n");	
  }
  printf("\n");
  return 0;
}
