
#include <hip/hip_runtime.h>
#include <stdio.h>
#define BLOCK_SIZE 128
#define BLOCK_SIZE_F 128.0

__global__
void sumRedKernel(float *A, int n){
  __shared__ float partialSum[BLOCK_SIZE*2];
  int i = (threadIdx.x + blockDim.x * blockIdx.x)*2;
  unsigned int t = threadIdx.x * 2;
  partialSum[t] = A[i];
  partialSum[t+1] = A[i+1];
  t = threadIdx.x;
  for(unsigned int stride = blockDim.x; stride > 0; stride /= 2){
    __syncthreads();
    if(t < stride && (t+stride) < n) partialSum[t] += partialSum[t+stride];
  }
  __syncthreads();
  if(threadIdx.x == 0){
    A[blockIdx.x] = partialSum[0];
  }
}

void sumRed(float* A, int n){
  int size = n*sizeof(float);
  float *d_A;

  hipMalloc((void **) &d_A, size);
  hipMemcpy(d_A,A,size,hipMemcpyHostToDevice);

  sumRedKernel<<<ceil(n/(BLOCK_SIZE_F*2)), BLOCK_SIZE>>>(d_A,n);
  
  hipMemcpy(A,d_A,size,hipMemcpyDeviceToHost);

  hipFree(d_A);
}


int main(){
  int n,i;
  float *h_A;
  scanf("%d", &n);
  h_A = (float*) malloc(n*sizeof(float));
  for(i = 0; i < n; i++){
    //scanf("%f", &h_A[i]);
    h_A[i] = 1;
  }
  while(n > 1){
     sumRed(h_A,n);
     n = ceil(n/(BLOCK_SIZE_F*2));
  }
  printf("%f", h_A[0]);
  printf("\n");
  return 0;
}
